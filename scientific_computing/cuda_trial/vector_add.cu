
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
  Cuda C program offload computation to GPU.
  Cuda provides C/C++ language extension and
  API for programming and managing GPUs.

  The __global__ specifier indicates a cuda function that runs on 
  (GPU).  Such function is called "kernels" and it is a global 
  function.

 */

/*
   nvcc -G -g -o ./run_cuda vector_add.cu
   nvcc -o ./run_cuda vector_add.cu -lineinfo
   nvprof ./run_cuda
   cuda-memcheck ./run_cuda
   compute-sanitizer ./run_cuda

  'lsb releae -r' 
        provides Ubuntu 20.04
  'nvidia-smi'
  'nvidia-smi --query-gpu=name --format=csv,noheader'
        NVIDIA GeForce RTX 3060 Laptop GPU
  
  sudo apt install nvidia-cuda-toolkit

  /user/include
  /user/include/c++/9
  /user/include/cuda-gdb

  CUDA organizes threads in groups named "threadblock" and the kernel can 
  launch multiple thread blocks, organized into a "grid" structuer.

  <<< M, T >>> which means grid of M thread blocks and each thread block has T
  parallel threads.

 */

/*
 function returns a void, has a pointer variable of data type float.
 out has values like 0x1234, *out has values like 1.1 and &out has
 values like 0xABCD
 */
__global__ void vector_add(float *out,
                float *first,
                float *second,
                int n)
{
    printf("Hello World From GPU!\n");
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid == 0)
        printf("tid %d threadIdx.x %d blockIdx.x %d blockDim.x %d gridDim.x %d\n", 
                tid,   threadIdx.x,   blockIdx.x,   blockDim.x,   gridDim.x);
    else
        printf("tid %d threadIdx.x %d blockIdx.x %d\n", 
                tid,   threadIdx.x,   blockIdx.x   );

    for (int cnt=0; cnt < n; cnt++)
    {
        out[cnt] = first[cnt] + second[cnt];
        printf("for Loop: tid %d threadIdx.x %d blockIdx.x %d\n", 
                          tid,   threadIdx.x,   blockIdx.x   );

    }
}

void print_pointers(float *out,
            int n)
{
   for (int cnt=0; cnt < n; cnt++)
   {
    printf("%f ",out[cnt]);
   }
   printf("\n");
}

int main()
{

    float * host_firstArray, * host_secondArray, * host_outPut;
    float * cuda_firstArray, * cuda_secondArray, * cuda_outPut;
    int n = 8;

    // Allocate host memory 
    host_firstArray = (float *)malloc(8 * sizeof(float));
    host_secondArray = (float *)malloc(8 * sizeof(float));
    host_outPut = (float *)malloc(8 * sizeof(float));
    

    // Allocate device memory for a
    hipMalloc( (void**)&cuda_firstArray, sizeof(float) * 8 );
    hipMalloc( (void**)&cuda_secondArray, sizeof(float) * 8 );
    hipMalloc( (void**)&cuda_outPut, sizeof(float) * 8 );

    for (int cnt = 0; cnt < n; cnt++)
    {
        host_firstArray[cnt] = 1.1;
        host_secondArray[cnt] = 2.2;
    }

    // Transfer data from host to device memory
    hipMemcpy(cuda_firstArray, host_firstArray, sizeof(float) * 8, hipMemcpyHostToDevice);
    hipMemcpy(cuda_secondArray, host_secondArray, sizeof(float) * 8, hipMemcpyHostToDevice);

    //Cuda Launch.
    vector_add<<<1, 1>>>(cuda_outPut, 
                         cuda_firstArray, 
                         cuda_secondArray, 
                         n);
   
    hipDeviceSynchronize();
    printf("Hello World From CPU!\n");

    hipMemcpy(host_outPut, cuda_outPut, sizeof(float) * 8, hipMemcpyDeviceToHost);

    print_pointers(host_outPut, n);

    free(host_firstArray);
    free(host_secondArray);
    free(host_outPut);

    // Cleanup after kernel execution
    hipFree(cuda_firstArray);
    hipFree(cuda_secondArray);
    hipFree(cuda_outPut);
    return 0;
}
